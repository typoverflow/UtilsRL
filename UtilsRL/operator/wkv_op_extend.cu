#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#define MIN_VALUE (-1e38)

template <typename F>
__global__ void kernel_forward(
    const int B, 
    const int T, 
    const int C,
    const F *__restrict__ const _w, 
    const F *__restrict__ const _u, 
    const F *__restrict__ const _k, 
    const F *__restrict__ const _v,
    const F *__restrict__ const _h1, 
    const F *__restrict__ const _h2,
    const F *__restrict__ const _h3, 
    F *__restrict__ const _y, 
    F *__restrict__ const _oh1, 
    F *__restrict__ const _oh2, 
    F *__restrict__ const _oh3
){
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int _hist_offset = _b * C + _c; 

    F u = _u[_c];
    F w = _w[_c];
    F h1 = _h1[_hist_offset]; 
    F h2 = _h2[_hist_offset]; 
    F h3 = _h3[_hist_offset]; 
    const F *__restrict__ const k = _k + _offset;
    const F *__restrict__ const v = _v + _offset;
    F *__restrict__ const y = _y + _offset;

    // aa and bb are running sums divided by exp(pp) (to avoid overflow)
    F aa = h1, bb = h2, pp = h3;
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const F kk = k[ii];
        const F vv = v[ii];

        F ww = u + kk;
        F p = max(pp, ww);
        F e1 = exp(pp - p);
        F e2 = exp(ww - p);
        y[ii] = (e1 * aa + e2 * vv) / (e1 * bb + e2);
        
        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;
    }

    _oh1[_hist_offset] = aa; 
    _oh2[_hist_offset] = bb; 
    _oh3[_hist_offset] = pp; 
}

void cuda_forward(
    int B, int T, int C, 
    float *w, float *u, float *k, float *v, 
    float *h1, float *h2, float *h3, float *y, 
    float *oh1, float *oh2, float *oh3
) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, h1, h2, h3, y, oh1, oh2, oh3);
}


template <typename F>
__global__ void kernel_backward(
    const int B, 
    const int T, 
    const int C,
    const F *__restrict__ const _w, 
    const F *__restrict__ const _u, 
    const F *__restrict__ const _k, 
    const F *__restrict__ const _v,
    const F *__restrict__ const _h1, 
    const F *__restrict__ const _h2,
    const F *__restrict__ const _h3, 
    const F *__restrict__ const _y, 
    const F *__restrict__ const _gy,
    F *__restrict__ const _gw, 
    F *__restrict__ const _gu, 
    F *__restrict__ const _gk, 
    F *__restrict__ const _gv
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int _hist_offset = _b * C + _c; 

    F u = _u[_c];
    F w = _w[_c];
    F h1 = _h1[_hist_offset]; 
    F h2 = _h2[_hist_offset]; 
    F h3 = _h3[_hist_offset]; 
    const F *__restrict__ const k = _k + _offset;
    const F *__restrict__ const v = _v + _offset;
    const F *__restrict__ const y = _y + _offset;
    const F *__restrict__ const gy = _gy + _offset;
    F *__restrict__ const gk = _gk + _offset;
    F *__restrict__ const gv = _gv + _offset;

    F q[Tmax], r[Tmax];

    F gw = 0, gu = 0, aa = h1, bb = h2, ga = 0, gb = 0, pp = h3;
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const F kk = k[ii];
        const F vv = v[ii];
        const F yy = y[ii];

        F ww = u + kk;
        F p = max(pp, ww);
        F e1 = exp(pp - p);
        F e2 = exp(ww - p);
        const F qq = gy[ii] / (e1 * bb + e2);
        gw += (ga - gb * yy) * e1 * qq;
        gu += (vv - yy) * e2 * qq;
        q[i] = qq;
        r[i] = ww - p;

        ww = w + pp;
        p = max(ww, kk);
        e1 = exp(ww - p);
        e2 = exp(kk - p);
        ga = e1 * (aa + ga);
        gb = e1 * (bb + gb);
        aa = e1 * aa + e2 * vv;
        bb = e1 * bb + e2;
        pp = p;
    }
    const int _offsetBC = _b * C + _c;
    _gw[_offsetBC] = gw * _w[_c]; // multiply by w because of w -> -exp(w) in python forward()
    _gu[_offsetBC] = gu;

    aa = h1, bb = h2, pp = h3;
    for (int i = T - 1; i >= 0; i--) {
        const int ii = i * C;
        const F kk = k[ii];
        const F vv = v[ii];
        const F yy = y[ii];
        const F qq = q[i];
        const F rr = r[i];

        F e1 = qq * exp(rr);
        F e2 = exp(kk + pp);
        gk[ii] = e1 * (vv - yy) + e2 * (aa * vv + bb);
        gv[ii] = e1 + e2 * aa;

        const F ww = w + pp;
        const F www = rr - u - kk;
        const F p = max(ww, www);
        e1 = exp(ww - p);
        e2 = qq * exp(www - p);
        aa = e1 * aa + e2;
        bb = e1 * bb - e2 * yy;
        pp = p;
    }
}

void cuda_backward(
    int B, int T, int C, 
    float *w, float *u, float *k, float *v,
    float *h1, float *h2, float *h3, float *y, 
    float *gy, float *gw, float *gu, float *gk, float *gv
) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_backward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, h1, h2, h3, y, gy, gw, gu, gk, gv);
}